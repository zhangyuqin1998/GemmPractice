#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "utils.h"

class GemmCublas : public GemmBase {
 public:
  using GemmBase::GemmBase;  // 继承基类的构造函数

  void LaunchKernel(const half *d_A, const half *d_B, float *d_C, uint64_t m,
                    uint64_t n, uint64_t k) override {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasGemmEx(
        handle,
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        n,
        m,
        k,
        &alpha,
        d_B,
        HIP_R_16F,
        k,
        d_A,
        HIP_R_16F,
        k,
        &beta,
        d_C,
        HIP_R_32F,
        n,
        HIP_R_32F,
        HIPBLAS_GEMM_DEFAULT
    );

  }
};

int main() {
  GemmCublas gemm("GemmCublas");
  // mxnxk
  // gemm.RunProfile(32, 32, 16);
  gemm.RunProfile(128, 128, 128);
  gemm.RunProfile(128, 256, 256);
  gemm.RunProfile(256, 512, 512);
  gemm.RunProfile(512, 256, 512);
  gemm.RunProfile(512, 512, 512);
  gemm.RunProfile(512, 512, 1024);
  gemm.RunProfile(512, 512, 2048);
  gemm.RunProfile(1024, 1024, 2048);
  gemm.RunProfile(1024, 256, 2048);
  gemm.RunProfile(256, 1024, 2048);
  gemm.RunProfile(2048, 2048, 2048);
  gemm.RunProfile(4096, 4096, 4096);

  return 0;
}
